// A simple example to demonstrate speedup by using cuda.


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays in a single thread.
__global__ void add(int n, float* x, float* y) {
  for (int i = 0; i < n; ++i) {
    y[i] += x[i];
  }
}

// Kernel function to add the elements in one block with many threads.
__global__ void add_threads(int n, float* x, float* y) {
  int index = threadIdx.x;  // the index of the current thread within its block.
  int stride = blockDim.x;  // the number of threads in the block
  for (int i = index; i < n; i += stride) {
    y[i] += x[i];
  }
}

// Kernel function to add the elements in multiple blocks with many threads.
__global__ void add_block_threads(int n, float* x, float* y) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] += x[i];
  }
}

int main() {
  int N = 1<<20;
  float* x;
  float* y;

  // Allocate unified memory - accessible from cpu to gpu
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // Initialize x and y arrays on the host
  for (int i = 0; i < N; ++i) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run the kernel on 1M elements on the GPU.
  add<<<1, 1>>>(N, x, y);  // one block and one thread

  // Run the kernel with one block and 256 threads.
  add_threads<<<1, 256>>>(N, x, y);

  // Run the kernel with many blocks and threads.
  int block_size = 256;
  int num_blocks = (N  + block_size - 1)/ block_size;
  add_block_threads<<<num_blocks, block_size>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; ++i) {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  }
  std::cout << "Max error: " << maxError << '\n';

  // Free memory
  hipFree(x);
  hipFree(y);
}
